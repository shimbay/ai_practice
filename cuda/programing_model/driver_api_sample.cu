#include <cstdio>
#include <cstring>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CUDA_CALL(call)                                                        \
  do {                                                                         \
    hipError_t res = (call);                                                     \
    if (res != hipSuccess) {                                                 \
      const char *errStr;                                                      \
      hipDrvGetErrorString(res, &errStr);                                          \
      fprintf(stderr,                                                          \
              "[CUDA ERROR] %s (%d)\n  File: %s\n  Line: %d\n  Call: %s\n",    \
              errStr, res, __FILE__, __LINE__, #call);                         \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

// PTX 内核代码 (SM 8.6 Ampere 架构)
const char *ptxKernel = R"(
// PTX 版本声明 (必须与设备兼容)
.version 7.8             // 对应 CUDA 11.x 及更高版本
.target sm_86             // 指定 Ampere 架构 (SM 8.6)
.address_size 64          // 64位地址空间

// 向量加法内核定义
.visible .entry vectorAdd(
    // 参数列表 (指针用 .u64，标量用 .u32)
    .param .u64 _A,       // 输入数组 A 的全局内存指针
    .param .u64 _B,       // 输入数组 B 的全局内存指针
    .param .u64 _C,       // 输出数组 C 的全局内存指针
    .param .u32 _N        // 数组长度
) {
    // 寄存器声明
    .reg .pred  %p<2>;    // 谓词寄存器 (用于条件判断)
    .reg .b32   %r<8>;    // 32位整数寄存器
    .reg .b64   %rd<10>;  // 64位地址寄存器
    .reg .f32   %f<4>;    // 32位浮点寄存器

    // 1. 加载参数到寄存器
    ld.param.u64    %rd1, [_A];     // 加载指针 A
    ld.param.u64    %rd2, [_B];     // 加载指针 B
    ld.param.u64    %rd3, [_C];     // 加载指针 C
    ld.param.u32    %r1, [_N];      // 加载数组长度 N

    // 2. 计算全局线程ID (blockIdx.x * blockDim.x + threadIdx.x)
    mov.u32         %r2, %tid.x;    // threadIdx.x
    mov.u32         %r3, %ctaid.x;  // blockIdx.x
    mov.u32         %r4, %ntid.x;   // blockDim.x
    mad.lo.s32      %r5, %r3, %r4, %r2;  // 计算全局索引

    // 3. 检查数组越界 (if (idx >= N) return)
    setp.ge.u32     %p1, %r5, %r1;  // 比较 idx >= N
    @%p1 bra        L_END;          // 若为真则跳转到结束

    // 4. 计算数组元素地址 (ptr = base + idx * sizeof(float))
    mul.wide.s32    %rd4, %r5, 4;   // idx * 4 (float占4字节)
    add.s64         %rd5, %rd1, %rd4;  // A + offset
    add.s64         %rd6, %rd2, %rd4;  // B + offset
    add.s64         %rd7, %rd3, %rd4;  // C + offset

    // 5. 从全局内存加载数据
    ld.global.f32   %f1, [%rd5];    // 加载 A[idx]
    ld.global.f32   %f2, [%rd6];    // 加载 B[idx]

    // 6. 执行加法运算
    add.f32         %f3, %f1, %f2;  // C[idx] = A[idx] + B[idx]

    // 7. 存储结果到全局内存
    st.global.f32   [%rd7], %f3;    // 存储 C[idx]

L_END:
    ret;                            // 内核结束
}
)";

int main() {
  hipDevice_t cuDevice, cuDevice1;
  hipCtx_t cuContext, cuContext1;
  hipModule_t cuModule;
  hipFunction_t cuFunction;
  hipStream_t cuStream;
  hipEvent_t cuStartEvent, cuStopEvent;
  int N = 1024;
  size_t bytes = N * sizeof(float);

  // === 1. 初始化 CUDA 驱动 ===
  CUDA_CALL(hipInit(0));

  // === 2. 获取设备并创建上下文 ===
  CUDA_CALL(hipDeviceGet(&cuDevice, 0));
  CUDA_CALL(hipCtxCreate(&cuContext, 0, cuDevice));

  CUDA_CALL(hipDeviceGet(&cuDevice1, 1));
  CUDA_CALL(hipCtxCreate(&cuContext1, 0, cuDevice1));

  int canAccessPeer;
  hipDeviceCanAccessPeer(&canAccessPeer, 0, 1);
  printf("can access result: %d\n", canAccessPeer);
  hipDeviceCanAccessPeer(&canAccessPeer, 1, 0);
  printf("can access result: %d\n", canAccessPeer);

  printf("\n=======================\n");
  int smCount, warpSize, maxThreadsPerSM, maxThreadsPerBlock;
  CUDA_CALL(hipDeviceGetAttribute(
      &smCount, hipDeviceAttributeMultiprocessorCount, cuDevice));
  CUDA_CALL(
      hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, cuDevice));
  CUDA_CALL(hipDeviceGetAttribute(
      &maxThreadsPerSM, hipDeviceAttributeMaxThreadsPerMultiProcessor,
      cuDevice));
  CUDA_CALL(hipDeviceGetAttribute(&maxThreadsPerBlock,
                                 hipDeviceAttributeMaxThreadsPerBlock,
                                 cuDevice));
  printf("Stream Multiprocessor num: %d\n", smCount);
  printf("Warp num per SM: %d\n", warpSize);
  printf("Max threads per SM: %d\n", maxThreadsPerSM);
  printf("Max threads per Block: %d\n", maxThreadsPerBlock);

  int maxGridDimX, maxGridDimY, maxGridDimZ;
  hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX,
                       cuDevice);
  hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY,
                       cuDevice);
  hipDeviceGetAttribute(&maxGridDimZ, hipDeviceAttributeMaxGridDimZ,
                       cuDevice);
  printf("Max Grid Dimensions: (%d, %d, %d)\n", maxGridDimX, maxGridDimY,
         maxGridDimZ);

  printf("=======================\n\n");

  // === 3. 创建 Stream 和 Event ===
  CUDA_CALL(hipStreamCreateWithFlags(&cuStream, hipStreamNonBlocking));
  CUDA_CALL(hipEventCreateWithFlags(&cuStartEvent, hipEventDefault));
  CUDA_CALL(hipEventCreateWithFlags(&cuStopEvent, hipEventDefault));

  // === 4. 加载 PTX 模块并获取内核 ===
  CUDA_CALL(hipModuleLoadData(&cuModule, ptxKernel));
  CUDA_CALL(hipModuleGetFunction(&cuFunction, cuModule, "vectorAdd"));

  // === 5. 分配主机和设备内存 ===
  float *h_A = (float *)malloc(bytes);
  float *h_B = (float *)malloc(bytes);
  float *h_C = (float *)malloc(bytes);
  for (int i = 0; i < N; i++) {
    h_A[i] = i;
    h_B[i] = i * 2;
  }

  hipDeviceptr_t d_A, d_B, d_C;
  CUDA_CALL(hipMalloc(&d_A, bytes));
  CUDA_CALL(hipMalloc(&d_B, bytes));
  CUDA_CALL(hipMalloc(&d_C, bytes));

  // === 6. 异步数据传输（主机→设备）===
  CUDA_CALL(hipMemcpyHtoDAsync(d_A, h_A, bytes, cuStream));
  CUDA_CALL(hipMemcpyHtoDAsync(d_B, h_B, bytes, cuStream));

  // === 7. 记录启动事件并执行内核 ===
  CUDA_CALL(hipEventRecord(cuStartEvent, cuStream));

  void *args[] = {&d_A, &d_B, &d_C, &N};
  CUDA_CALL(hipModuleLaunchKernel(cuFunction, (N + 255) / 256, 1, 1, // Grid 维度
                           256, 1, 1,                         // Block 维度
                           0,         // 共享内存大小
                           cuStream,  // 绑定 Stream
                           args, NULL // 内核参数
                           ));

  // === 8. 记录结束事件并异步回传数据 ===
  CUDA_CALL(hipEventRecord(cuStopEvent, cuStream));
  CUDA_CALL(hipMemcpyDtoHAsync(h_C, d_C, bytes, cuStream));

  // === 9. 同步 Stream 并计算耗时 ===
  CUDA_CALL(hipStreamSynchronize(cuStream));
  float elapsedMs;
  CUDA_CALL(hipEventElapsedTime(&elapsedMs, cuStartEvent, cuStopEvent));

  // === 10. 验证结果 ===
  printf("VectorAdd completed in %.3f ms\n", elapsedMs);
  for (int i = 0; i < 20; i++) {
    printf("C[%d] = %.1f (A=%.1f + B=%.1f)\n", i, h_C[i], h_A[i], h_B[i]);
  }

  return 0;
}
